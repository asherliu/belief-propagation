#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <>

extern "C" {
#include "../bnf-parser/expression.h"
#include "../bnf-parser/Parser.h"
#include "../bnf-parser/Lexer.h"
#include "../bnf-xml-parser/xml-expression.h"
}

int yyparse(struct expression ** expr, yyscan_t scanner);

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

__device__
void init_message_buffer_cuda(float * buffer, float * node_states, unsigned int num_variables, unsigned int node_index){
    unsigned int j;

    for(j = 0; j < num_variables; ++j){
        buffer[j] = node_states[MAX_STATES * node_index + j];
    }

}

__device__
void combine_message_cuda(float * dest, float * edge_messages, unsigned int length, unsigned int offset){
    unsigned int i;

    for(i = 0; i < length; ++i){
        if(edge_messages[offset + i] == edge_messages[offset + i]){
            dest[i] = dest[i] * edge_messages[offset + i];
        }
    }
}

__device__
void read_incoming_messages_cuda(float * message_buffer, float * previous_messages, unsigned int * dest_nodes_to_edges,
                                 unsigned int current_num_edges,
                            unsigned int num_vertices, unsigned int num_variables, unsigned int idx){
    unsigned int start_index, end_index, j, edge_index;

    start_index = dest_nodes_to_edges[idx];
    if(idx + 1 >= num_vertices){
        end_index = num_vertices + current_num_edges;
    }
    else{
        end_index = dest_nodes_to_edges[idx + 1];
    }
    for(j = start_index; j < end_index; ++j){
        edge_index = dest_nodes_to_edges[j];

        combine_message_cuda(message_buffer, previous_messages, num_variables, MAX_STATES * edge_index);
    }
}

__device__
void send_message_for_edge_cuda(float * buffer, unsigned int edge_index, float * joint_probabilities,
                                float * edge_messages, unsigned int * x_dim, unsigned int * y_dim){
    unsigned int i, j, num_src, num_dest;
    float sum;

    num_src = x_dim[edge_index];
    num_dest = y_dim[edge_index];

    sum = 0.0;
    for(i = 0; i < num_src; ++i){
        edge_messages[edge_index * MAX_STATES + i] = 0.0;
        for(j = 0; j < num_dest; ++j){
            edge_messages[edge_index * MAX_STATES + i] += joint_probabilities[MAX_STATES * MAX_STATES * edge_index + MAX_STATES * i + j] * buffer[j];
        }
        sum += edge_messages[edge_index * MAX_STATES + i];
    }
    if(sum <= 0.0){
        sum = 1.0;
    }
    for(i = 0; i < num_src; ++i){
        edge_messages[edge_index * MAX_STATES + i] = edge_messages[edge_index * MAX_STATES + i] / sum;
    }
}

__device__
void send_message_for_node_cuda(float * message_buffer, unsigned int current_num_edges,
                                float * joint_probabilities, float * current_edge_messages,
                                unsigned int * src_nodes_to_edges,
                                unsigned int * edges_x_dim, unsigned int * edges_y_dim,
                                unsigned int num_vertices, unsigned int idx){
    unsigned int start_index, end_index, j, edge_index;

    start_index = src_nodes_to_edges[idx];
    if(idx + 1 >= num_vertices){
        end_index = num_vertices + current_num_edges;
    }
    else{
        end_index = src_nodes_to_edges[idx + 1];
    }

    for(j = start_index; j < end_index; ++j){
        edge_index = src_nodes_to_edges[j];
        send_message_for_edge_cuda(message_buffer, edge_index, joint_probabilities, current_edge_messages, edges_x_dim, edges_y_dim);
    }
}

__device__
void marginalize_node(unsigned int * node_num_vars, float * node_states, unsigned int idx,
                        float * current_edges_messages, unsigned int * dest_nodes_to_edges,
                      unsigned int num_vertices, unsigned int num_edges){
    unsigned int i, num_variables, start_index, end_index, edge_index;
    char has_incoming;
    float sum;

    has_incoming = 0;

    num_variables = node_num_vars[idx];

    float new_message[MAX_STATES];

    for(i = 0; i < num_variables; ++i){
        new_message[i] = 1.0;
    }

    start_index = dest_nodes_to_edges[idx];
    if(idx + 1 >= num_vertices){
        end_index = num_vertices + num_edges;
    }
    else{
        end_index = dest_nodes_to_edges[idx + 1];
    }

    for(i = start_index; i < end_index; ++i){
        edge_index = dest_nodes_to_edges[i];

        combine_message_cuda(new_message, current_edges_messages, num_variables, MAX_STATES * edge_index);
        has_incoming = 1;
    }
    if(has_incoming == 1){
        for(i = 0; i < num_variables; ++i){
            node_states[MAX_STATES * idx + i] = new_message[i];
        }
    }
    sum = 0.0;
    for(i = 0; i < num_variables; ++i){
        sum += node_states[MAX_STATES * idx + i];
    }
    if(sum <= 0.0){
        sum = 1.0;
    }
    for(i = 0; i < num_variables; ++i){
        node_states[MAX_STATES * idx + i] = node_states[MAX_STATES * idx + i] / sum;
    }
}

__global__
void loopy_propagate_main_loop(unsigned int num_vertices, unsigned int num_edges,
                                unsigned int * node_num_vars, float * node_messages,
                               float * joint_probabilities,
                               float * previous_edge_messages, float * current_edge_messages,
                               unsigned int * src_nodes_to_edges, unsigned int * dest_nodes_to_edges,
                               unsigned int * edges_x_dim, unsigned int * edges_y_dim){
    unsigned int idx, num_variables;
    float message_buffer[MAX_STATES];

    idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < num_vertices){
        num_variables = node_num_vars[idx];

        init_message_buffer_cuda(message_buffer, node_messages, num_variables, idx);
        __syncthreads();

        read_incoming_messages_cuda(message_buffer, previous_edge_messages, dest_nodes_to_edges, num_edges, num_vertices, num_variables, idx);
        __syncthreads();

        send_message_for_node_cuda(message_buffer, num_edges, joint_probabilities, current_edge_messages, src_nodes_to_edges, edges_x_dim, edges_y_dim, num_vertices, idx);
        __syncthreads();

        marginalize_node(node_num_vars, node_messages, idx, current_edge_messages, dest_nodes_to_edges, num_vertices, num_edges);
    }

    __syncthreads();
}

__device__
float calculate_local_delta(unsigned int i, float * previous_messages, float * current_messages, unsigned int * x_dim){
    float delta, diff;
    unsigned int k, num_messages;

    delta = 0.0;
    num_messages = x_dim[i];

    for(k = 0; k < num_messages; ++k){
        diff = previous_messages[MAX_STATES * i + k] - current_messages[MAX_STATES * i + k];
        if(diff != diff){
            diff = 0.0;
        }
        delta += fabs(diff);
    }

    return delta;
}

__global__
void calculate_delta(float * previous_messages, float * current_messages, float * delta, float * delta_array,
                     unsigned int * x_dim,
                     unsigned int num_edges){
    extern __shared__ float shared_delta[];
    unsigned int tid, idx, i, s;

    tid = threadIdx.x;
    idx = blockIdx.x*blockDim.x + threadIdx.x;
    i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    if(idx < num_edges){
        delta_array[idx] = calculate_local_delta(idx, previous_messages, current_messages, x_dim);
    }
    __syncthreads();

    float my_delta = (i < num_edges) ? delta_array[i] : 0;

    if(i + BLOCK_SIZE < num_edges){
        my_delta += delta_array[i + BLOCK_SIZE];
    }

    shared_delta[tid] = my_delta;
    __syncthreads();

    // do reduction in shared mememory
    for(s= blockDim.x / 2; s > 32; s>>=1){
        if(tid < s){
            shared_delta[tid] = my_delta = my_delta + shared_delta[tid + s];
        }

        __syncthreads();
    }

#if (__CUDA_ARCH__ >= 300)
    if(tid < 32){
        //fetch final intermediate sum from second warp
        if(BLOCK_SIZE >= 64){
            my_delta += shared_delta[tid + 32];
        }
        for(s = WARP_SIZE/2; s > 0; s /= 2){
            my_delta += __shfl_down(my_delta, s);
        }
    }
#else
    if((BLOCK_SIZE >= 64) && (tid < 32)){
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 32];
    }
    __syncthreads();
    if((BLOCK_SIZE >= 32) && (tid < 16)){
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 16];
    }
    __syncthreads();
    if((BLOCK_SIZE >= 16) && (tid < 8)){
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 8];
    }
    __syncthreads();
    if((BLOCK_SIZE >= 8) && (tid < 4)){
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 4];
    }
    __syncthreads();
    if((BLOCK_SIZE >= 4) && (tid < 2)){
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 2];
    }
    __syncthreads();
    if((BLOCK_SIZE >= 2) && (tid < 1)){
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 1];
    }
    __syncthreads();
#endif
    if(tid == 0) {
        *delta = my_delta;
    }
}

__global__
void calculate_delta_6(float * previous_messages, float * current_messages, float * delta, float * delta_array,
                       unsigned int * x_dim,
                       unsigned int num_edges, char n_is_pow_2, unsigned int warp_size) {
    extern __shared__ float shared_delta[];

    unsigned int offset;
    // perform first level of reduce
    // reading from global memory, writing to shared memory
    unsigned int idx =  blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    unsigned int grid_size = blockDim.x * 2 * gridDim.x;

    if(idx < num_edges){
        delta_array[i] = calculate_local_delta(idx, previous_messages, current_messages, x_dim);
    }
    __syncthreads();

    float my_delta = 0.0;

    while (i < num_edges) {
        my_delta = delta_array[i];

        // ensure we don't read out of bounds
        if (n_is_pow_2 || i + blockDim.x < num_edges) {
            my_delta += delta_array[i];
        }

        i += grid_size;
    }

    //each thread puts its local sum into shared memory
    shared_delta[tid] = my_delta;
    __syncthreads();

    // do reduction in shared mem
    if ((blockDim.x >= 512) && (tid < 256)) {
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 256];
    }
    __syncthreads();
    if ((blockDim.x >= 256) && (tid < 128)) {
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 128];
    }
    __syncthreads();
    if ((blockDim.x >= 128) && (tid < 64)) {
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 64];
    }
    __syncthreads();

#if (__CUDA_ARCH__ >= 300)
    if( tid < 32){
        // fetch final intermediate sum from 2nd warp
        if(blockDim.x >= 64){
            my_delta += shared_delta[tid + 32];
        }
        for(offset = warp_size/2; offset > 0; offset /= 2 ){
            my_delta += __shfl_down(my_delta, offset);
        }
    }
#else
    // fully unroll reduction within a single warp
    if ((blockDim.x >= 64) && (tid < 32)) {
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 32];
    }
    __syncthreads();

    if ((blockDim.x >= 32) && (tid < 16)) {
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 16];
    }
    __syncthreads();

    if((blockDim.x >= 16) && (tid < 8)){
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 8];
    }
    __syncthreads();

    if((blockDim.x >= 8) && (tid < 4)){
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 4];
    }
    __syncthreads();

    if((blockDim.x >= 4) && (tid < 2)){
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 2];
    }
    __syncthreads();

    if((blockDim.x >= 2) && (tid < 1)){
        shared_delta[tid] = my_delta = my_delta + shared_delta[tid + 1];
    }
    __syncthreads();

#endif
    //write result for this block to global mem
    if(tid == 0){
        *delta = my_delta;
    }
}

__global__
void calculate_delta_simple(float * previous_messages, float * current_messages,
                            float * delta, float * delta_array, unsigned int * x_dim,
                            unsigned int num_edges) {
    extern __shared__ float shared_delta[];
    unsigned int tid, idx, i, s;

    tid = threadIdx.x;
    idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_edges) {
        delta_array[idx] = calculate_local_delta(idx, previous_messages, current_messages, x_dim);
    }
    __syncthreads();

    shared_delta[tid] = (idx < num_edges) ? delta_array[idx] : 0;

    __syncthreads();

    // do reduction in shared mem
    for(s = 1; s < blockDim.x; s *= 2){
        i = 2 * s * tid;
        if( i < blockDim.x ) {
            shared_delta[i] += shared_delta[i + s];
        }

        __syncthreads();
    }

    //write result for this block to global mem
    if(tid == 0){
        *delta = shared_delta[0];
    }
}

static void prepare_unsigned_int_text(texture<unsigned int, hipTextureType1D, hipReadModeElementType> * tex){
    tex->addressMode[0] = hipAddressModeWrap;
    tex->addressMode[1] = hipAddressModeWrap;
    tex->filterMode = hipFilterModePoint;
    tex->normalized = 1;
}

unsigned int loopy_propagate_until_cuda(Graph_t graph, float convergence, unsigned int max_iterations){
    unsigned int i, j, num_iter, num_vertices, num_edges;
    float * delta;
    float * delta_array;
    float previous_delta, host_delta;
    char is_pow_2;

    float * edges_joint_probabilities;

    float * current_messages;
    float * previous_messages;
    float * temp;

    float * node_states;
    unsigned int * node_num_vars;

    hipError_t err;

    host_delta = 0.0;

    struct hipChannelFormatDesc channel_desc_unsigned_int = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);

    unsigned int * dest_node_to_edges;
    unsigned int * src_node_to_edges;
    unsigned int * edges_x_dim;
    unsigned int * edges_y_dim;

    num_vertices = graph->current_num_vertices;
    num_edges = graph->current_num_edges;

    /*printf("Before=====");
    print_edges(graph);
    print_nodes(graph);*/


    is_pow_2 = num_vertices % 2 == 0;

    // allocate data
    CUDA_CHECK_RETURN(hipMalloc((void **)&dest_node_to_edges, sizeof(unsigned int) * (graph->current_num_edges + graph->current_num_vertices)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&src_node_to_edges, sizeof(unsigned int) * (graph->current_num_edges + graph->current_num_vertices)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&edges_x_dim, sizeof(unsigned int) * graph->current_num_edges));
    CUDA_CHECK_RETURN(hipMalloc((void **)&edges_y_dim, sizeof(unsigned int) * graph->current_num_edges));

    CUDA_CHECK_RETURN(hipMalloc((void **)&edges_joint_probabilities, sizeof(float) * MAX_STATES * MAX_STATES * graph->current_num_edges));

    CUDA_CHECK_RETURN(hipMalloc((void **)&current_messages, sizeof(float) * MAX_STATES * graph->current_num_edges));
    CUDA_CHECK_RETURN(hipMalloc((void **)&previous_messages, sizeof(float) * MAX_STATES * graph->current_num_edges));

    CUDA_CHECK_RETURN(hipMalloc((void **)&node_states, sizeof(float) * MAX_STATES * graph->current_num_vertices));
    CUDA_CHECK_RETURN(hipMalloc((void **)&node_num_vars, sizeof(unsigned int) * graph->current_num_vertices));

    CUDA_CHECK_RETURN(hipMalloc((void **)&delta, sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **)&delta_array, sizeof(float) * num_vertices));

    // copy data
    CUDA_CHECK_RETURN(hipMemcpy(edges_joint_probabilities, graph->edges_joint_probabilities, sizeof(float) * MAX_STATES * MAX_STATES * graph->current_num_edges, hipMemcpyHostToDevice ));

    CUDA_CHECK_RETURN(hipMemcpy(current_messages, graph->edges_messages, sizeof(float) * MAX_STATES * graph->current_num_edges, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(current_messages, graph->last_edges_messages, sizeof(float) * MAX_STATES * graph->current_num_edges, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(node_num_vars, graph->node_num_vars, sizeof(unsigned int) * graph->current_num_vertices, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(node_states, graph->node_states, sizeof(float) * MAX_STATES * graph->current_num_vertices, hipMemcpyHostToDevice));

    CUDA_CHECK_RETURN(hipMemcpy(dest_node_to_edges, graph->dest_nodes_to_edges, sizeof(unsigned int) * (num_edges + num_vertices), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(src_node_to_edges, graph->src_nodes_to_edges, sizeof(unsigned int) * (num_edges + num_vertices), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(edges_x_dim, graph->edges_x_dim, sizeof(unsigned int) * num_edges, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(edges_y_dim, graph->edges_y_dim, sizeof(unsigned int) * num_edges, hipMemcpyHostToDevice));

    const int blockCount = (num_edges + BLOCK_SIZE - 1)/ BLOCK_SIZE;
    num_iter = 0;

    dim3 dimReduceBlock(BLOCK_SIZE, 1, 1);
    dim3 dimReduceGrid(blockCount, 1, 1);
    int reduceSmemSize = (BLOCK_SIZE <= 32) ? 2 * BLOCK_SIZE * sizeof(float) : BLOCK_SIZE * sizeof(float);

    for(i = 0; i < max_iterations; i+= BATCH_SIZE){
        for(j = 0; j < BATCH_SIZE; ++j) {
            loopy_propagate_main_loop<<<blockCount, BLOCK_SIZE >>>(num_vertices, num_edges, node_num_vars, node_states, edges_joint_probabilities, previous_messages, current_messages, src_node_to_edges, dest_node_to_edges, edges_x_dim, edges_y_dim);
            err = hipGetLastError();
            if (err != hipSuccess) {
                fprintf(stderr, "Error: %s\n", hipGetErrorString(err));
                exit(-1);
            }
            //swap pointers
            temp = current_messages;
            current_messages = previous_messages;
            previous_messages = temp;
            num_iter++;
        }
        calculate_delta_6<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(previous_messages, current_messages, delta, delta_array, edges_x_dim, num_edges, is_pow_2, WARP_SIZE);
        //calculate_delta<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(previous_messages, current_messages, delta, delta_array, edges_x_dim, num_edges);
        //calculate_delta_simple<<<dimReduceGrid, dimReduceBlock, reduceSmemSize>>>(previous_messages, current_messages, delta, delta_array, edges_x_dim, num_edges);
        err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Error: %s\n", hipGetErrorString(err));
        }
        CUDA_CHECK_RETURN(hipMemcpy(&host_delta, delta, sizeof(float), hipMemcpyDeviceToHost));
     //   printf("Current delta: %f\n", host_delta);

        if(host_delta < convergence || fabs(host_delta - previous_delta) < convergence){
            break;
        }
        previous_delta = host_delta;
    }

    // copy data back
    CUDA_CHECK_RETURN(hipMemcpy(graph->node_states, node_states, sizeof(float) * MAX_STATES * num_vertices, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipMemcpy(graph->edges_messages, current_messages, sizeof(float) * MAX_STATES * num_edges, hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipMemcpy(graph->last_edges_messages, previous_messages, sizeof(float) * MAX_STATES * num_edges, hipMemcpyDeviceToHost));

    CUDA_CHECK_RETURN(hipFree(dest_node_to_edges));
    CUDA_CHECK_RETURN(hipFree(src_node_to_edges));
    CUDA_CHECK_RETURN(hipFree(edges_x_dim));
    CUDA_CHECK_RETURN(hipFree(edges_y_dim));

    CUDA_CHECK_RETURN(hipFree(edges_joint_probabilities));

    CUDA_CHECK_RETURN(hipFree(current_messages));
    CUDA_CHECK_RETURN(hipFree(previous_messages));

    CUDA_CHECK_RETURN(hipFree(node_states));
    CUDA_CHECK_RETURN(hipFree(node_num_vars));

    CUDA_CHECK_RETURN(hipFree(delta));
    CUDA_CHECK_RETURN(hipFree(delta_array));

    /*printf("After=====");
    print_nodes(graph);
    print_edges(graph);*/

    return num_iter;
}

void test_ast(const char * expr)
{
    struct expression * expression;
    yyscan_t scanner;
    YY_BUFFER_STATE state;

    assert(yylex_init(&scanner) == 0);

    assert(scanner != NULL);
    assert(strlen(expr) > 0);

    state = yy_scan_string(expr, scanner);

    assert(yyparse(&expression, scanner) == 0);
    yy_delete_buffer(state, scanner);
    yylex_destroy(scanner);

    assert(expression != NULL);

    delete_expression(expression);
}

void test_file(const char * file_path)
{
    struct expression * expression;
    yyscan_t scanner;
    YY_BUFFER_STATE state;
    FILE * in;

    assert(yylex_init(&scanner) == 0);

    in = fopen(file_path, "r");

    yyset_in(in, scanner);

    assert(yyparse(&expression, scanner) == 0);
    //yy_delete_buffer(state, scanner);
    yylex_destroy(scanner);

    fclose(in);

    assert(expression != NULL);

    delete_expression(expression);
}

void test_parse_file(char * file_name){
    unsigned int i;
    struct expression * expression;
    yyscan_t scanner;
    YY_BUFFER_STATE state;
    FILE * in;
    Graph_t graph;
    clock_t start, end;
    double time_elapsed;

    assert(yylex_init(&scanner) == 0);

    in = fopen(file_name, "r");

    yyset_in(in, scanner);

    assert(yyparse(&expression, scanner) == 0);
    //yy_delete_buffer(state, scanner);
    yylex_destroy(scanner);

    fclose(in);

    assert(expression != NULL);

    graph = build_graph(expression);
    //print_nodes(graph);
    //print_edges(graph);

    set_up_src_nodes_to_edges(graph);
    set_up_dest_nodes_to_edges(graph);

    start = clock();
    init_levels_to_nodes(graph);
    //print_levels_to_nodes(graph);

    propagate_using_levels_start(graph);
    for(i = 1; i < graph->num_levels - 1; ++i){
        propagate_using_levels(graph, i);
    }
    reset_visited(graph);
    for(i = graph->num_levels - 1; i > 0; --i){
        propagate_using_levels(graph, i);
    }

    marginalize(graph);
    end = clock();

    time_elapsed = (double)(end - start) / CLOCKS_PER_SEC;
    printf("%s,regular,%d,%d,%lf\n", file_name, graph->current_num_vertices, graph->current_num_edges, time_elapsed);

    //print_nodes(graph);

    assert(graph != NULL);

    delete_expression(expression);

    graph_destroy(graph);
}

void test_loopy_belief_propagation(char * file_name){
    struct expression * expression;
    yyscan_t scanner;
    YY_BUFFER_STATE state;
    FILE * in;
    Graph_t graph;
    clock_t start, end;
    double time_elapsed;

    assert(yylex_init(&scanner) == 0);

    in = fopen(file_name, "r");

    yyset_in(in, scanner);

    assert(yyparse(&expression, scanner) == 0);
    //yy_delete_buffer(state, scanner);
    yylex_destroy(scanner);

    fclose(in);

    assert(expression != NULL);

    graph = build_graph(expression);
    assert(graph != NULL);
    //print_nodes(graph);
    //print_edges(graph);

    set_up_src_nodes_to_edges(graph);
    set_up_dest_nodes_to_edges(graph);

    start = clock();
    init_previous_edge(graph);

    loopy_propagate_until_cuda(graph, PRECISION, NUM_ITERATIONS);
    end = clock();

    time_elapsed = (double)(end - start)/CLOCKS_PER_SEC;
    //print_nodes(graph);
    printf("%s,loopy,%d,%d,%lf\n", file_name, graph->current_num_vertices, graph->current_num_edges, time_elapsed);

    delete_expression(expression);

    graph_destroy(graph);
}

struct expression * parse_file(const char * file_name){
    struct expression * expression;
    yyscan_t scanner;
    YY_BUFFER_STATE state;
    FILE * in;

    assert(yylex_init(&scanner) == 0);

    in = fopen(file_name, "r");

    yyset_in(in, scanner);

    assert(yyparse(&expression, scanner) == 0);
    //yy_delete_buffer(state, scanner);
    yylex_destroy(scanner);

    fclose(in);

    assert(expression != NULL);

    return expression;
}

void run_test_belief_propagation(struct expression * expression, const char * file_name){
    Graph_t graph;
    clock_t start, end;
    double time_elapsed;
    unsigned int i;

    graph = build_graph(expression);
    assert(graph != NULL);
    //print_nodes(graph);
    //print_edges(graph);

    set_up_src_nodes_to_edges(graph);
    set_up_dest_nodes_to_edges(graph);
    calculate_diameter(graph);

    start = clock();
    init_levels_to_nodes(graph);
    //print_levels_to_nodes(graph);

    propagate_using_levels_start(graph);
    for(i = 1; i < graph->num_levels - 1; ++i){
        propagate_using_levels(graph, i);
    }
    reset_visited(graph);
    for(i = graph->num_levels - 1; i > 0; --i){
        propagate_using_levels(graph, i);
    }

    marginalize(graph);
    end = clock();

    time_elapsed = (double)(end - start) / CLOCKS_PER_SEC;
    printf("%s,regular,%d,%d,%d,2,%lf\n", file_name, graph->current_num_vertices, graph->current_num_edges, graph->diameter, time_elapsed);

    graph_destroy(graph);
}

void run_test_loopy_belief_propagation(struct expression * expression, const char * file_name, FILE * out){
    Graph_t graph;
    clock_t start, end;
    double time_elapsed;
    unsigned int num_iterations;

    graph = build_graph(expression);
    assert(graph != NULL);
    //print_nodes(graph);
    //print_edges(graph);

    set_up_src_nodes_to_edges(graph);
    set_up_dest_nodes_to_edges(graph);
    //calculate_diameter(graph);

    start = clock();
    init_previous_edge(graph);

    num_iterations = loopy_propagate_until_cuda(graph, PRECISION, NUM_ITERATIONS);
    end = clock();

    time_elapsed = (double)(end - start)/CLOCKS_PER_SEC;
    //print_nodes(graph);
    fprintf(out, "%s,loopy,%d,%d,%d,%d,%lf\n", file_name, graph->current_num_vertices, graph->current_num_edges, graph->diameter, num_iterations, time_elapsed);
    fflush(out);

    graph_destroy(graph);
}

void run_tests_with_file(const char * file_name, unsigned int num_iterations, FILE * out){
    unsigned int i;
    struct expression * expr;

    expr = parse_file(file_name);
    for(i = 0; i < num_iterations; ++i){
        run_test_belief_propagation(expr, file_name);
    }

    for(i = 0; i < num_iterations; ++i){
        run_test_loopy_belief_propagation(expr, file_name, out);
    }

    delete_expression(expr);
}

void run_tests_with_xml_file(const char * file_name, unsigned int num_iterations, FILE * out){
    unsigned int i;
    struct expression * expr;

    expr = parse_xml_file(file_name);
    /*for(i = 0; i < num_iterations; ++i){
        run_test_belief_propagation(expr, file_name);
    }*/

    for(i = 0; i < num_iterations; ++i){
        run_test_loopy_belief_propagation(expr, file_name, out);
    }

    delete_expression(expr);
}

int main(void)
{
/*
	extern int yydebug;
	yydebug = 1;
/*
	struct expression * expression = NULL;
	const char test[] = "// Bayesian Network in the Interchange Format\n// Produced by BayesianNetworks package in JavaBayes\n// Output created Sun Nov 02 17:49:49 GMT+00:00 1997\n// Bayesian network \nnetwork \"Dog-Problem\" { //5 variables and 5 probability distributions\nproperty \"credal-set constant-density-bounded 1.1\" ;\n}variable  \"light-on\" { //2 values\ntype discrete[2] {  \"true\"  \"false\" };\nproperty \"position = (218, 195)\" ;\n}\nvariable  \"bowel-problem\" { //2 values\ntype discrete[2] {  \"true\"  \"false\" };\nproperty \"position = (335, 99)\" ;\n}";
	test_ast(test);

  	test_parse_file("dog.bif");
	test_parse_file("alarm.bif");

	test_parse_file("very_large/andes.bif");
	test_loopy_belief_propagation("very_large/andes.bif");

	test_parse_file("Diabetes.bif");
	test_loopy_belief_propagation("Diabetes.bif");
*/
	//test_loopy_belief_propagation("../benchmark_files/dog.bif");
	//test_loopy_belief_propagation("../benchmark_files/alarm.bif");

    //test_file("dog.bif");
    //test_file("alarm.bif");

    /*expression = read_file("alarm.bif");

    assert(expression != NULL);

    delete_expression(expression);*/

    FILE * out = fopen("cuda_benchmark.csv", "w");
    fprintf(out, "File Name,Propagation Type,Number of Nodes,Number of Edges,Diameter,Number of Iterations,BP Run Time(s)\n");
    fflush(out);

	/*run_tests_with_file("../benchmark_files/small/asia.bif", 1);
	run_tests_with_file("../benchmark_files/small/cancer.bif", 1);
	run_tests_with_file("../benchmark_files/small/earthquake.bif", 1);
	run_tests_with_file("../benchmark_files/small/sachs.bif", 1);
	run_tests_with_file("../benchmark_files/small/survey.bif", 1);
/*
	run_tests_with_file("../benchmark_files/medium/alarm.bif", 1);
	run_tests_with_file("../benchmark_files/medium/barley.bif", 1);
	//run_tests_with_file("../benchmark_files/medium/child.bif", 1);
	run_tests_with_file("../benchmark_files/medium/hailfinder.bif", 1);
	run_tests_with_file("../benchmark_files/medium/insurance.bif", 1);
	run_tests_with_file("../benchmark_files/medium/mildew.bif", 1);
	run_tests_with_file("../benchmark_files/medium/water.bif", 1);

	run_tests_with_file("../benchmark_files/large/hepar2.bif", 1);
	run_tests_with_file("../benchmark_files/large/win95pts.bif", 1);

    run_tests_with_file("../benchmark_files/very_large/andes.bif", 1);
    run_tests_with_file("../benchmark_files/very_large/diabetes.bif", 1);
    run_tests_with_file("../benchmark_files/very_large/link.bif", 1);
    run_tests_with_file("../benchmark_files/very_large/munin1.bif", 1);
    run_tests_with_file("../benchmark_files/very_large/munin2.bif", 1);
    run_tests_with_file("../benchmark_files/very_large/munin3.bif", 1);
    run_tests_with_file("../benchmark_files/very_large/munin4.bif", 1);
	//run_tests_with_file("../benchmark_files/very_large/munin.bif", 1);
	run_tests_with_file("../benchmark_files/very_large/pathfinder.bif", 1);
    run_tests_with_file("../benchmark_files/very_large/pigs.bif", 1);

    run_tests_with_xml_file("../benchmark_files/xml/bf_1000_2000_1.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_1000_2000_2.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_1000_2000_3.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_2000_4000_1.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_2000_4000_2.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_2000_4000_3.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_5000_10000_1.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_5000_10000_2.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_5000_10000_3.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_10000_20000_1.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_10000_20000_2.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_10000_20000_3.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_12000_24000_1.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_12000_24000_2.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_12000_24000_3.xml", 1);*/

    /*run_tests_with_xml_file("../benchmark_files/xml/bf_15000_30000_1.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_15000_30000_2.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_15000_30000_3.xml", 1);

    run_tests_with_xml_file("../benchmark_files/xml/bf_20000_40000_1.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_20000_40000_2.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_20000_40000_3.xml", 1);

    run_tests_with_xml_file("../benchmark_files/xml/bf_25000_50000_1.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_25000_50000_2.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_25000_50000_3.xml", 1);

    run_tests_with_xml_file("../benchmark_files/xml/bf_30000_60000_1.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_30000_60000_2.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_30000_60000_3.xml", 1);*/

    /*run_tests_with_xml_file("../benchmark_files/xml/bf_40000_80000_1.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_40000_80000_2.xml", 1);
    run_tests_with_xml_file("../benchmark_files/xml/bf_40000_80000_3.xml", 1);

    run_tests_with_xml_file("../benchmark_files/xml/bf_80000_160000_2.xml", 1);*/

    run_tests_with_xml_file("../benchmark_files/xml2/10_20.xml", 1, out);
    run_tests_with_xml_file("../benchmark_files/xml2/100_200.xml", 1, out);
    run_tests_with_xml_file("../benchmark_files/xml2/1000_2000.xml", 1, out);
    run_tests_with_xml_file("../benchmark_files/xml2/10000_20000.xml", 1, out);
    run_tests_with_xml_file("../benchmark_files/xml2/100000_200000.xml", 1, out);
    run_tests_with_xml_file("../benchmark_files/xml2/200000_400000.xml", 1, out);
    //run_tests_with_xml_file("../benchmark_files/xml2/300000_600000.xml", 1, out);
    run_tests_with_xml_file("../benchmark_files/xml2/400000_800000.xml", 1, out);
    //run_tests_with_xml_file("../benchmark_files/xml2/500000_1000000.xml", 1, out);
    run_tests_with_xml_file("../benchmark_files/xml2/600000_1200000.xml", 1, out);
    //run_tests_with_xml_file("../benchmark_files/xml2/700000_1400000.xml", 1, out);
    run_tests_with_xml_file("../benchmark_files/xml2/800000_1600000.xml", 1, out);
    //run_tests_with_xml_file("../benchmark_files/xml2/900000_1800000.xml", 1, out);
    run_tests_with_xml_file("../benchmark_files/xml2/1000000_2000000.xml", 1, out);
    //run_tests_with_xml_file("../benchmark_files/xml2/10000000_20000000.xml", 1, out);

    return 0;
}


/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned int line, const char *statement, hipError_t err)
{
    if (err == hipSuccess)
        return;
    printf("%s returned %s (%d) at %s:%d\n", statement, hipGetErrorString(err), err, file, line);
    exit (1);
}

